
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
}

int main(void)
{
    hello_from_gpu<<<1, 1>>>();
    //同步等待，要GPU跑完才能走
    hipDeviceSynchronize();
    return 0;
}